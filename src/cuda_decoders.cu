#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "cuda_decoders.h"

#include "g722decoder.h"

#define checkCudaErrors(val) CheckErrors( (val), __FILE__, __LINE__)

void CheckErrors(hipError_t cuda_error, const char* const file, const int line)
{
    if (cuda_error != hipSuccess)
	{
        std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(cuda_error) << std::endl;
        exit(1);
    }		
}

__global__ void CudaKernelG711aToPcm(unsigned char* d_alaw_data_ptr, short int* d_pcm_data_ptr)
{
    unsigned int idx = 160*(threadIdx.x + blockDim.x * blockIdx.x);
	
	short int quantization_value;
	short int quantization_segment;
	unsigned char alaw_data;

	for (int k=0; k<160; k++)
	{
	    alaw_data = d_alaw_data_ptr[idx+k];	
		alaw_data^=0x55;

		quantization_value= (alaw_data & (0xf)) << 4;
		quantization_segment = ((unsigned)alaw_data & (0x70)) >> (4);
		switch (quantization_segment)
		{
		case 0: 
			quantization_value+=(0x0008);
			break;
		case 1:
			quantization_value+=(0x0108);
			break;
		default:
			quantization_value+=(0x0108);
			quantization_value <<= (quantization_segment-1);
		};
		
		d_pcm_data_ptr[idx+k]=((alaw_data & (0x80))?quantization_value:-quantization_value);
	}
}

// memory coalesced version of alaw to pcm conversion
__global__ void CudaKernelG711aToPcmCM(unsigned char* d_alaw_data_ptr, short int* d_pcm_data_ptr)
{
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int total_threads = blockDim.x * gridDim.x;
	
	short int quantization_value;
	short int quantization_segment;
	unsigned char alaw_data;

	for (int k=0; k<160; k++, idx+=total_threads)
	{
	    alaw_data = d_alaw_data_ptr[idx];	
		alaw_data^=0x55;

		quantization_value= (alaw_data & (0xf)) << 4;
		quantization_segment = ((unsigned)alaw_data & (0x70)) >> (4);
		switch (quantization_segment)
		{
		case 0: 
			quantization_value+=(0x0008);
			break;
		case 1:
			quantization_value+=(0x0108);
			break;
		default:
			quantization_value+=(0x0108);
			quantization_value <<= (quantization_segment-1);
		};
		
		d_pcm_data_ptr[idx]=((alaw_data & (0x80))?quantization_value:-quantization_value);
	}
}

__global__ void CudaKernelG711uToPcm(unsigned char* d_ulaw_data_ptr, short int* d_pcm_data_ptr)
{
    unsigned int idx = 160*(threadIdx.x + blockDim.x * blockIdx.x);
	
	short int quantization_value;
	short int quantization_segment;
	unsigned char ulaw_data;

	for (int k=0; k<160; k++)
	{
		ulaw_data=~(d_ulaw_data_ptr[idx+k]);

		quantization_value= (ulaw_data & (0xf)) << 4;
		quantization_segment = ((unsigned)ulaw_data & (0x70)) >> (4);

		quantization_value += 0x0084;
		quantization_value <<= quantization_segment;

		quantization_value-=(32);
	
		d_pcm_data_ptr[idx+k]=((ulaw_data & (0x80))?quantization_value:-quantization_value);
	}
}

__global__ void CudaKernelG711uToPcmCM(unsigned char* d_ulaw_data_ptr, short int* d_pcm_data_ptr)
{
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int total_threads = blockDim.x * gridDim.x;
		
	short int quantization_value;
	short int quantization_segment;
	unsigned char ulaw_data;

	for (int k=0; k<160; k++, idx+=total_threads)
	{
		ulaw_data=~(d_ulaw_data_ptr[idx]);

		quantization_value= (ulaw_data & (0xf)) << 4;
		quantization_segment = ((unsigned)ulaw_data & (0x70)) >> (4);

		quantization_value += 0x0084;
		quantization_value <<= quantization_segment;

		quantization_value-=(32);
	
		d_pcm_data_ptr[idx]=((ulaw_data & (0x80))?quantization_value:-quantization_value);
	}
}

__global__ void CudaKernelG722ToPcm(unsigned char* d_g722_data_ptr, short int* d_pcm_data_ptr)
{
    unsigned int idx = 160*(threadIdx.x + blockDim.x * blockIdx.x);
	
	unsigned char g722_data;

	for (int k=0; k<160; k++)
	{
		g722_data=d_g722_data_ptr[idx+k];

		d_pcm_data_ptr[idx+k]=0;
	}
}

__device__ short int CudaConvertLongToShort(int in_value)
{
	if (in_value > 32767)
		return 32767;
	else if (in_value < -32768)
		return -32768;
	else
		return (short)in_value;
}

__global__ void CudaKernelG722ToPcmCM(unsigned char* d_g722_data_ptr, short int* d_pcm_data_ptr, int* d_band_data_ptr, int* d_g722_consts_ptr, unsigned int no_of_data)
{
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int total_threads = blockDim.x * gridDim.x;
		
	unsigned char g722_data;
	
	int number_of_chunks = no_of_data/160;
	if (idx >= number_of_chunks)
	    return;
		
	// pointers for constants, maybe copy these to shared mem
	int* wl = d_g722_consts_ptr;
	int* rl42 = d_g722_consts_ptr + 8;
	int* ilb = rl42 + 16;
	int* qm4 = ilb + 32;
	int* qm6 = qm4 + 16;

	
	// copy band data to local variables
	int band_s = d_band_data_ptr[idx];
	idx += number_of_chunks;
	int band_sp = d_band_data_ptr[idx];
	idx += number_of_chunks;
	int band_sz = d_band_data_ptr[idx];
	idx += number_of_chunks;
	
	int band_r[3], band_a[3], band_ap[3], band_p[3];
	for(int k=0; k<3; k++, idx+=number_of_chunks)
	{
	    band_r[k] = d_band_data_ptr[idx];
		band_a[k] = d_band_data_ptr[idx + 3*number_of_chunks];
		band_ap[k] = d_band_data_ptr[idx + 6*number_of_chunks];
	    band_p[k] = d_band_data_ptr[idx + 9*number_of_chunks];
	}
	idx += 9*number_of_chunks;
	
	int band_d[7], band_b[7], band_bp[7], band_sg[7];
	for(int k=0; k<7; k++, idx+=number_of_chunks)
	{
	    band_d[k] = d_band_data_ptr[idx];
		band_b[k] = d_band_data_ptr[idx + 7*number_of_chunks];
		band_bp[k] = d_band_data_ptr[idx + 14*number_of_chunks];
	    band_sg[k] = d_band_data_ptr[idx + 21*number_of_chunks];
	}
	idx += 21*number_of_chunks;
	
	int band_nb = d_band_data_ptr[idx];
	idx += number_of_chunks;
	int band_det = d_band_data_ptr[idx];
	//band_det=32;
	
	int dlowt;
	int rlow;
	int wd1;
	int wd2;
	int wd3;
	
	idx = threadIdx.x + blockDim.x * blockIdx.x;
	for (int k=0; k<160; k++, idx+=number_of_chunks)
	{
		g722_data=d_g722_data_ptr[idx];

	    wd1 = g722_data & 0x3F;
	    wd2 = qm6[wd1];
	    wd1 >>= 2;

	    /********************** Block 5 *******************/
	    // INVQBL (ITU page 43), compute quantized difference signal for the decoder output in the lower sub-band
	    wd2 = (band_det * wd2) >> 15;
	    // RECONS ( ITU page 41), compute reconstructed signal for the adaptive predictor
	    rlow = band_s + wd2;
	
	    /********************** Block 6 ********************/
	    // LIMIT (ITU page 44), limit the output reconstructed signal
	    if (rlow > 16383)
		    rlow = 16383;
	    else if (rlow < -16384)
		    rlow = -16384;

	    /********************** Block 2 ***********************/	
	    // INVQAL (ITU page 37), compute the quantized differences signal for the adaptive predictor in the lower sub-band
	    wd2 = qm4[wd1];
	    dlowt = (band_det * wd2) >> 15;

	    /********************** Block 3 ************************/
	    // LOGSCL (ITU page 38), update the logarithmic quantizer scale factor in the lower sub-band
	    wd2 = rl42[wd1];
	    wd1 = (band_nb * 127) >> 7;
	    wd1 += wl[wd2];
	    if (wd1 < 0)
		    wd1 = 0;
	    else if (wd1 > 18432)
		    wd1 = 18432;
	    band_nb = wd1;

	    // SCALEL (ITU page 38), compute the quantizer scale factor in the lower sub-band 
	    wd1 = (band_nb >> 6) & 31;
	    wd2 = 8 - (band_nb >> 11);
	    wd3 = (wd2 < 0)	 ?  (ilb[wd1] << -wd2)	:  (ilb[wd1] >> wd2);
	    band_det = wd3 << 2;

	    /********************** Block 4 **************************/

	    // RECONS (ITU page 41), compute reconstructed signal for the adaptive predictor
	    band_d[0] = dlowt;
	    band_r[0] = CudaConvertLongToShort(band_s + dlowt);

	    // PARREC (ITU page 40), compute partially reconstructed signal
	    band_p[0] = CudaConvertLongToShort(band_sz + dlowt);

	    // UPPOL2 (ITU page 41), update second predictor coefficient
	    int i;  // loop variable
	    for (i = 0;	 i < 3;	 i++)
		    band_sg[i] = band_p[i] >> 15;
	    wd1 = CudaConvertLongToShort(band_a[1] << 2);

	    wd2 = (band_sg[0] == band_sg[1])	?  -wd1	 :  wd1;
	    if (wd2 > 32767)
		    wd2 = 32767;
	    wd3 = (band_sg[0] == band_sg[2])	?  128	:  -128;
	    wd3 += (wd2 >> 7);
	    wd3 += (band_a[2]*32512) >> 15;
	    if (wd3 > 12288)
	    	wd3 = 12288;
	    else if (wd3 < -12288)
		    wd3 = -12288;
	    band_ap[2] = wd3;

	    // UPPOL1 (ITU page 42), update first predictor coefficient
	    band_sg[0] = band_p[0] >> 15;
	    band_sg[1] = band_p[1] >> 15;
	    wd1 = (band_sg[0] == band_sg[1])	?  192	:  -192;
	    wd2 = (band_a[1]*32640) >> 15;

	    band_ap[1] = CudaConvertLongToShort(wd1 + wd2);
	    wd3 = CudaConvertLongToShort(15360 - band_ap[2]);
	    if (band_ap[1] > wd3)
		    band_ap[1] = wd3;
	    else if (band_ap[1] < -wd3)
		    band_ap[1] = -wd3;

	    // UPZERO (ITU page 41), update sixth order predictor coefficients
	    wd1 = (dlowt == 0)  ?  0  :  128;
	    band_sg[0] = dlowt >> 15;
	    for (i = 1;	 i < 7;	 i++)
	    {
		    band_sg[i] = band_d[i] >> 15;
		    wd2 = (band_sg[i] == band_sg[0])  ?  wd1  :  -wd1;
		    wd3 = (band_b[i]*32640) >> 15;
		    band_bp[i] = CudaConvertLongToShort(wd2 + wd3);
	    }

	    // DELAYA (ITU page 38), memory block delay 
	    for (i = 6;	 i > 0;	 i--)
	    {
		    band_d[i] = band_d[i - 1];
		    band_b[i] = band_bp[i];
	    }

	    for (i = 2;	 i > 0;	 i--)
	    {
		    band_r[i] = band_r[i - 1];
		    band_p[i] = band_p[i - 1];
		    band_a[i] = band_ap[i];
	    }

	    // FILTEP (ITU page 43), compute predictor output signal, poles
	    wd1 = CudaConvertLongToShort(band_r[1] + band_r[1]);
	    wd1 = (band_a[1]*wd1) >> 15;
	    wd2 = CudaConvertLongToShort(band_r[2] + band_r[2]);
	    wd2 = (band_a[2]*wd2) >> 15;
	    band_sp = CudaConvertLongToShort(wd1 + wd2);

	    // FILTEZ (ITU page 42), compute predictor output signal, zeros
	    band_sz = 0;
	    for (i = 6;	 i > 0;	 i--)
	    {
		    wd1 = CudaConvertLongToShort(band_d[i] + band_d[i]);
		    band_sz += (band_b[i]*wd1) >> 15;
	    }
	    band_sz = CudaConvertLongToShort(band_sz);

	    // PREDIC (ITU page 43), compute predictor output value
	    band_s = CudaConvertLongToShort(band_sp + band_sz);

		d_pcm_data_ptr[idx]=(short int)rlow;
	}
	
	
	
	// copy local variables back to band data
	idx = threadIdx.x + blockDim.x * blockIdx.x;
	
	d_band_data_ptr[idx] = band_s;
	idx += number_of_chunks;
	band_sp = d_band_data_ptr[idx]= band_sp;
	idx += number_of_chunks;
	d_band_data_ptr[idx] = band_sz;
	idx += number_of_chunks;
	
	for(int k=0; k<3; k++, idx+=number_of_chunks)
	{
	    d_band_data_ptr[idx] = band_r[k];
		d_band_data_ptr[idx + 3*number_of_chunks] = band_a[k];
		d_band_data_ptr[idx + 6*number_of_chunks] = band_ap[k];
	    d_band_data_ptr[idx + 9*number_of_chunks] = band_p[k];
	}
	idx += 9*number_of_chunks;
	
	for(int k=0; k<7; k++, idx+=number_of_chunks)
	{
	    d_band_data_ptr[idx] = band_d[k];
		d_band_data_ptr[idx + 7*number_of_chunks] = band_b[k];
		d_band_data_ptr[idx + 14*number_of_chunks] = band_bp[k];
	    d_band_data_ptr[idx + 21*number_of_chunks] = band_sg[k];
	}
	idx += 21*number_of_chunks;
	
	d_band_data_ptr[idx] = band_nb;
	idx += number_of_chunks;
	d_band_data_ptr[idx] = band_det;
	idx += number_of_chunks;
}

int CudaG711aToPcm(const unsigned char* alaw_data_ptr, unsigned int no_of_data, short int* pcm_data_ptr)
{
    dim3 block_dim(THREAD_PER_BLOCK, 1, 1);
	dim3 grid_dim(ceil(no_of_data/((float)(block_dim.x * 160))), 1, 1);
	
	unsigned int size_of_alaw_data = no_of_data * sizeof(unsigned char);
	unsigned int size_of_pcm_data = no_of_data * sizeof(short int);
	
	unsigned int no_of_d_data = grid_dim.x * block_dim.x * 160;
	unsigned int size_of_d_alaw_data = no_of_d_data * sizeof(unsigned char);
	unsigned int size_of_d_pcm_data = no_of_d_data * sizeof(short int);
	
	unsigned char* d_alaw_data_ptr = NULL;
    hipMalloc((void**)&d_alaw_data_ptr, size_of_d_alaw_data);
	checkCudaErrors(hipGetLastError());
	
	short int* d_pcm_data_ptr = NULL;
    hipMalloc((void**)&d_pcm_data_ptr, size_of_d_pcm_data);
	checkCudaErrors(hipGetLastError());
	
    hipMemcpy(d_alaw_data_ptr, alaw_data_ptr, size_of_alaw_data, hipMemcpyHostToDevice);
    checkCudaErrors(hipGetLastError());
	
    // launch kernel here
	CudaKernelG711aToPcmCM <<< grid_dim, block_dim >>> (d_alaw_data_ptr, d_pcm_data_ptr);
    checkCudaErrors(hipGetLastError());
	
    hipMemcpy(pcm_data_ptr, d_pcm_data_ptr, size_of_pcm_data, hipMemcpyDeviceToHost);
    checkCudaErrors(hipGetLastError());
		
    hipFree(d_alaw_data_ptr);
	checkCudaErrors(hipGetLastError());
    
	hipFree(d_pcm_data_ptr);
    checkCudaErrors(hipGetLastError());	

    return 0;
}

int CudaG711uToPcm(const unsigned char* ulaw_data_ptr, unsigned int no_of_data, short int* pcm_data_ptr)
{
    dim3 block_dim(THREAD_PER_BLOCK, 1, 1);
	dim3 grid_dim(ceil(no_of_data/((float)(block_dim.x * 160))), 1, 1);
	
	unsigned int size_of_ulaw_data = no_of_data * sizeof(unsigned char);
	unsigned int size_of_pcm_data = no_of_data * sizeof(short int);
	
	unsigned int no_of_d_data = grid_dim.x * block_dim.x * 160;
	unsigned int size_of_d_ulaw_data = no_of_d_data * sizeof(unsigned char);
	unsigned int size_of_d_pcm_data = no_of_d_data * sizeof(short int);
	
	unsigned char* d_ulaw_data_ptr = NULL;
    hipMalloc((void**)&d_ulaw_data_ptr, size_of_d_ulaw_data);
	checkCudaErrors(hipGetLastError());
	
	short int* d_pcm_data_ptr = NULL;
    hipMalloc((void**)&d_pcm_data_ptr, size_of_d_pcm_data);
	checkCudaErrors(hipGetLastError());
	
    hipMemcpy(d_ulaw_data_ptr, ulaw_data_ptr, size_of_ulaw_data, hipMemcpyHostToDevice);
    checkCudaErrors(hipGetLastError());
	
    // launch kernel here
	CudaKernelG711uToPcmCM <<< grid_dim, block_dim >>> (d_ulaw_data_ptr, d_pcm_data_ptr);
    checkCudaErrors(hipGetLastError());
	
    hipMemcpy(pcm_data_ptr, d_pcm_data_ptr, size_of_pcm_data, hipMemcpyDeviceToHost);
    checkCudaErrors(hipGetLastError());
		
    hipFree(d_ulaw_data_ptr);
	checkCudaErrors(hipGetLastError());
    
	hipFree(d_pcm_data_ptr);
    checkCudaErrors(hipGetLastError());	

    return 0;
}

int CudaG722ToPcm(const unsigned char* g722_data_ptr, int* band_data_ptr, unsigned int no_of_data, short int* pcm_data_ptr)
{
    dim3 block_dim(THREAD_PER_BLOCK, 1, 1);
	dim3 grid_dim(ceil(no_of_data/((float)(block_dim.x * 160))), 1, 1);
	
	unsigned int size_of_g722_data = no_of_data * sizeof(unsigned char);
	unsigned int size_of_pcm_data = no_of_data * sizeof(short int);
	
	unsigned int no_of_d_data = grid_dim.x * block_dim.x * 160;
	unsigned int size_of_d_g722_data = no_of_d_data * sizeof(unsigned char);
	unsigned int size_of_d_pcm_data = no_of_d_data * sizeof(short int);
	
	unsigned char* d_g722_data_ptr = NULL;
    hipMalloc((void**)&d_g722_data_ptr, size_of_d_g722_data);
	checkCudaErrors(hipGetLastError());
	
	short int* d_pcm_data_ptr = NULL;
    hipMalloc((void**)&d_pcm_data_ptr, size_of_d_pcm_data);
	checkCudaErrors(hipGetLastError());
	
    hipMemcpy(d_g722_data_ptr, g722_data_ptr, size_of_g722_data, hipMemcpyHostToDevice);
    checkCudaErrors(hipGetLastError());
	
	// calculate space for band, 45 integers per thread
	unsigned int number_of_d_band_data = grid_dim.x * block_dim.x;
	unsigned int size_of_d_band_data = number_of_d_band_data * 45 * sizeof(int);
	unsigned int number_of_band_data = no_of_data/160;
	unsigned int size_of_band_data = number_of_band_data * 45 * sizeof(int);
	
	//std::cout << "size of band data " << size_of_d_band_data << std::endl;
	
	int* d_band_data_ptr = NULL;
    hipMalloc((void**)&d_band_data_ptr, size_of_d_band_data);
	checkCudaErrors(hipGetLastError());
	hipMemcpy(d_band_data_ptr, band_data_ptr, size_of_band_data, hipMemcpyHostToDevice);
	checkCudaErrors(hipGetLastError());	
	
	unsigned int size_of_d_g722_consts = sizeof(g722_consts);
	int* d_g722_consts_ptr = NULL;
    hipMalloc((void**)&d_g722_consts_ptr, size_of_d_g722_consts);
	checkCudaErrors(hipGetLastError());
	hipMemcpy(d_g722_consts_ptr, g722_consts, size_of_d_g722_consts, hipMemcpyHostToDevice);
	checkCudaErrors(hipGetLastError());	
 
	
    // launch kernel here
	CudaKernelG722ToPcmCM <<< grid_dim, block_dim >>> (d_g722_data_ptr, d_pcm_data_ptr, d_band_data_ptr, d_g722_consts_ptr, no_of_data);
    checkCudaErrors(hipGetLastError());
	
	//std::cout << "host pcm data size   : " << size_of_pcm_data << std::endl;
	//std::cout << "device pcm data size : " << size_of_d_pcm_data << std::endl;
	//std::cout << "no of data           : " << no_of_data << std::endl;
	//std::cout << "number of threads    : " << no_of_d_data << std::endl;
	
    hipMemcpy(pcm_data_ptr, d_pcm_data_ptr, size_of_pcm_data, hipMemcpyDeviceToHost);
    checkCudaErrors(hipGetLastError());
	
	hipMemcpy(band_data_ptr, d_band_data_ptr, size_of_band_data, hipMemcpyDeviceToHost);
	checkCudaErrors(hipGetLastError());
	
	hipFree(d_g722_consts_ptr);
	checkCudaErrors(hipGetLastError());
	
	hipFree(d_band_data_ptr);
	checkCudaErrors(hipGetLastError());
		
    hipFree(d_g722_data_ptr);
	checkCudaErrors(hipGetLastError());
    
	hipFree(d_pcm_data_ptr);
    checkCudaErrors(hipGetLastError());	

    return 0;
}

void CudaGpuInitialize()
{
    unsigned int size_of_d_dummy_data = 1000000;
    unsigned char* d_dummy_data_ptr = NULL;

    //unsigned int device_count = hipGetDeviceCount();
    //checkCudaErrors(hipGetLastError()); 
    
    //std::cout << "device count : " << device_count << std::endl
	
    hipMalloc((void**)&d_dummy_data_ptr, size_of_d_dummy_data);
    checkCudaErrors(hipGetLastError());
	
    hipMemset((void*)d_dummy_data_ptr, 0, size_of_d_dummy_data);
    checkCudaErrors(hipGetLastError());
	
    hipFree(d_dummy_data_ptr);
    checkCudaErrors(hipGetLastError());
}
